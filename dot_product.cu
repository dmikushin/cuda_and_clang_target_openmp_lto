
#include <hip/hip_runtime.h>
#if 0
// Will not work:
//
// nvlink error   : Undefined reference to '_Z10dotProductPfS_iii' in 'matmul.nvcc.o' 
inline __attribute__((always_inline))
#endif
__device__ float dotProduct(float *A, float *B, int row, int col, int N) {
    float sum = 0.0f;
    for(int k = 0; k < N; ++k) {
        sum += A[row * N + k] * B[k * N + col];
    }
    return sum;
}

