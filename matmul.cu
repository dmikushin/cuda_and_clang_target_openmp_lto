
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <ctime>

using namespace std;

__device__ float dotProduct(float *A, float *B, int row, int col, int N);

__global__ void matMulKernel(float *A, float *B, float *C, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if(row < N && col < N) {
        C[row * N + col] = dotProduct(A, B, row, col, N);
    }
}

void matMulCPU(float *A, float *B, float *C, int N) {
    for(int i = 0; i < N; ++i) {
        for(int j = 0; j < N; ++j) {
            float sum = 0.0f;
            for(int k = 0; k < N; ++k) {
                sum += A[i * N + k] * B[k * N + j];
            }
            C[i * N + j] = sum;
        }
    }
}

bool checkResults(float *C, float *C_CPU, int N) {
    for(int i = 0; i < N*N; ++i) {
        if(abs(C[i] - C_CPU[i]) > 1e-3) {
            std::cerr << "Result mismatch @ i = " << i << " : " << C[i] << " != " << C_CPU[i] << std::endl;
	    return false;
        }
    }
    return true;
}

int main(int argc, char **argv) {
    int N = (argc > 1) ? atoi(argv[1]) : 1024;
    cout << "Matrix size: " << N << "x" << N << endl;

    // Initialize random seed
    srand(time(0));

    float *A = new float[N*N];
    float *B = new float[N*N];
    float *C_GPU, *C_CPU, *C;

    // Generate random matrices A and B
    for(int i = 0; i < N*N; ++i) {
        A[i] = rand() / float(RAND_MAX);
        B[i] = rand() / float(RAND_MAX);
    }

    hipMalloc(&C_GPU, N*N*sizeof(float));
    C_CPU = new float[N*N]();
    C = new float[N*N]();

    matMulCPU(A, B, C_CPU, N);

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((N + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (N + threadsPerBlock.y - 1) / threadsPerBlock.y);
    
    matMulKernel<<<numBlocks, threadsPerBlock>>>(A, B, C_GPU, N);

    hipMemcpy(C, C_GPU, N*N*sizeof(float), hipMemcpyDeviceToHost);

    if(checkResults(C, C_CPU, N)) {
        cout << "Result is correct!" << endl;
    } else {
        cout << "Result is incorrect." << endl;
    }

    hipFree(C_GPU);
    delete[] C_CPU;
    delete[] A;
    delete[] B;
    delete[] C;

    return 0;
}
